#include "hip/hip_runtime.h"
#include "cpu_reducer.h"

namespace byteps {
namespace common {

__global__ void sum_kernel(float* dst, const float* src, size_t len,
                           float alpha) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) dst[i] += src[i] * alpha;
}

__global__ void sum_kernel(float* dst, const float* src1, const float* src2,
                           size_t len, float alpha) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) dst[i] = src1[i] + src2[i] * alpha;
}

__global__ void sign_kernel(float* dst, const float* src, size_t len) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) dst[i] = signbit(src[i]);
}

__global__ void norm1_kernel(float* src, float* out, size_t len) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= len) return;

  float* data = src + blockIdx.x * blockDim.x;

  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (tid < stride) {
      data[tid] = abs(data[tid]) + abs(data[tid + stride]);
    }
    __syncthreads();
  }

  if (tid == 0) atomicAdd(out, data[0]);
}

int CpuReducer::sum(void* dst, const void* src, size_t len, int dtype,
                    float alpha) {
  hipMemcpy(dev_src1, src, len, hipMemcpyHostToDevice);

  sum_kernel<<<_block_per_grid, _thread_per_block>>>(dev_dst, dev_src1, len / 4,
                                                     alpha);

  hipMemcpy(dst, dev_dst, len, hipMemcpyDeviceToHost);
  return 0;
}

int CpuReducer::sum(void* dst, const void* src1, const void* src2, size_t len,
                    int dtype, float alpha) {
  hipMemcpy(dev_src1, src1, len, hipMemcpyHostToDevice);
  hipMemcpy(dev_src2, src2, len, hipMemcpyHostToDevice);

  sum_kernel<<<_block_per_grid, _thread_per_block>>>(dev_dst, dev_src1,
                                                     dev_src2, len / 4, alpha);

  hipMemcpy(dst, dev_dst, len, hipMemcpyDeviceToHost);
  return 0;
}

int CpuReducer::sign(void* dst, const void* src, size_t len, int dtype) {
  hipMemcpy(dev_src1, src, len, hipMemcpyHostToDevice);

  sign_kernel<<<_block_per_grid, _thread_per_block>>>(dev_dst, dev_src1,
                                                      len / 4);

  hipMemcpy(dst, dev_dst, len, hipMemcpyDeviceToHost);
  return 0;
}

float CpuReducer::norm1(const void* src, size_t len, int dtype) {
  hipMemcpy(dev_src1, src, len, hipMemcpyHostToDevice);

  norm1_kernel<<<_block_per_grid, _thread_per_block>>>(dev_src1, dev_scalar,
                                                       len / 4);

  float ret = 0.0;
  hipMemcpy(&ret, dev_scalar, 4, hipMemcpyDeviceToHost);
  return ret;
}

}  // namespace common
}  // namespace byteps
