#include "hip/hip_runtime.h"
#include "cpu_reducer.h"
#include "logging.h"

__global__ void sum_kernel(float* dst, const float* src, size_t len,
                           float alpha) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) dst[i] += src[i] * alpha;
}

__global__ void sum_kernel(float* dst, const float* src1, const float* src2,
                           size_t len, float alpha) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) dst[i] = src1[i] + src2[i] * alpha;
}

__global__ void sign_kernel(int* dst, const float* src, size_t len) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) dst[i] = signbit(src[i]);
}

__global__ void norm1_kernel(float* src, float* out, size_t len) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= len) return;

  float* data = src + blockIdx.x * blockDim.x;

  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (tid < stride) {
      data[tid] = abs(data[tid]) + abs(data[tid + stride]);
    }
    __syncthreads();
  }

  if (tid == 0) atomicAdd(out, data[0]);
}

namespace byteps {
namespace common {

int CpuReducer::sum(void* dev_dst, const void* dev_src, size_t len, int dtype,
                    float alpha) {
  sum_kernel<<<_block_per_grid, _thread_per_block, 0, *_stream>>>(
      reinterpret_cast<float*>(dev_dst),
      reinterpret_cast<float*>(const_cast<void*>(dev_src)), len / 4, alpha);
  return 0;
}

int CpuReducer::sum(void* dev_dst, const void* dev_src1, const void* dev_src2,
                    size_t len, int dtype, float alpha) {
  sum_kernel<<<_block_per_grid, _thread_per_block, 0, *_stream>>>(
      reinterpret_cast<float*>(dev_dst),
      reinterpret_cast<float*>(const_cast<void*>(dev_src1)),
      reinterpret_cast<float*>(const_cast<void*>(dev_src2)), len / 4, alpha);
  return 0;
}

int CpuReducer::sign(void* dev_dst, const void* dev_src, size_t len,
                     int dtype) {
  sign_kernel<<<_block_per_grid, _thread_per_block, 0, *_stream>>>(
      reinterpret_cast<int*>(dev_dst),
      reinterpret_cast<float*>(const_cast<void*>(dev_src)), len / 4);
  return len / 4;
}

int CpuReducer::norm1(void* dev_src, float* dev_out, size_t len, int dtype) {
  norm1_kernel<<<_block_per_grid, _thread_per_block, 0, *_stream>>>(
      reinterpret_cast<float*>(dev_src), dev_out, len / 4);

  return 0;
}

}  // namespace common
}  // namespace byteps
