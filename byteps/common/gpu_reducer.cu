#include "hip/hip_runtime.h"
#include "cpu_reducer.h"
#include "logging.h"

__global__ void sum_kernel(float* dst, const float* src, size_t len,
                           float alpha) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) dst[i] += src[i] * alpha;
}

__global__ void sum_kernel(float* dst, const float* src1, const float* src2,
                           size_t len, float alpha) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) dst[i] = src1[i] + src2[i] * alpha;
}

__global__ void sign_kernel(int* dst, const float* src, size_t len) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) dst[i] = signbit(src[i]);
}

__global__ void norm1_kernel(const float* src, float* out, size_t len) {
  // max size 16KB
  __shared__ float vec[1024];

  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  vec[tid] = (idx < len) ? src[idx] : 0;
  __syncthreads();

  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (tid < stride) {
      vec[tid] = abs(vec[tid]) + abs(vec[tid + stride]);
    }
    __syncthreads();
  }

  if (tid == 0) atomicAdd(out, vec[0]);
}

namespace byteps {
namespace common {
constexpr int BLOCK_PER_GRID = 1024;
// int CpuReducer::sum(void* dev_dst, const void* dev_src, size_t len, int
// dtype,
//                     float alpha) {
//   int thread_per_block = ((len/4) + BLOCK_PER_GRID) / BLOCK_PER_GRID;
//   sum_kernel<<<BLOCK_PER_GRID, thread_per_block, 0, *_stream>>>(
//       reinterpret_cast<float*>(dev_dst),
//       reinterpret_cast<const float*>(const_cast<void*>(dev_src)), len / 4,
//       alpha);
//   return 0;
// }

int CpuReducer::sum(void* dev_dst, const void* dev_src1, const void* dev_src2,
                    size_t len, int dtype, float alpha) {
  int thread_per_block = ((len / 4) + BLOCK_PER_GRID) / BLOCK_PER_GRID;
  sum_kernel<<<BLOCK_PER_GRID, thread_per_block, 0, *_stream>>>(
      reinterpret_cast<float*>(dev_dst),
      reinterpret_cast<const float*>(const_cast<void*>(dev_src1)),
      reinterpret_cast<const float*>(const_cast<void*>(dev_src2)), len / 4,
      alpha);
  return 0;
}

int CpuReducer::sign(void* dev_dst, const void* dev_src, size_t len,
                     int dtype) {
  int thread_per_block = ((len / 4) + BLOCK_PER_GRID) / BLOCK_PER_GRID;
  sign_kernel<<<BLOCK_PER_GRID, thread_per_block, 0, *_stream>>>(
      reinterpret_cast<int*>(dev_dst),
      reinterpret_cast<const float*>(const_cast<void*>(dev_src)), len / 4);
  return len / 4;
}

int CpuReducer::norm1(const void* dev_src, float* dev_out, size_t len,
                      int dtype) {
  int x = ((len / 4) + BLOCK_PER_GRID) / BLOCK_PER_GRID;
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  ++x;
  norm1_kernel<<<BLOCK_PER_GRID, x, 0, *_stream>>>(
      reinterpret_cast<const float*>(const_cast<void*>(dev_src)), dev_out,
      len / 4);
  return 0;
}

}  // namespace common
}  // namespace byteps
