#include "hip/hip_runtime.h"
#include <iostream>

#include "compressor/strategy/onebit.h"
#include "cpu_reducer.h"
#include "logging.h"

__global__ void sum_kernel(float* dst, const float* src, size_t len,
                           float alpha) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) dst[i] += src[i] * alpha;
}

__global__ void sum_kernel(float* dst, const float* src1, const float* src2,
                           size_t len, float alpha) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) dst[i] = src1[i] + src2[i] * alpha;
}

__global__ void sign_kernel(int* dst, const float* src, size_t len) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) dst[i] = signbit(src[i]);
}

__global__ void norm1_kernel(const float* src, float* dst, size_t len) {
  // max size 16KB
  __shared__ float vec[1024];
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  vec[tid] = (idx < len) ? src[idx] : 0;
  __syncthreads();

  for (int stride = (blockDim.x >> 1); stride > 0; stride >>= 1) {
    if (tid < stride) {
      vec[tid] = abs(vec[tid]) + abs(vec[tid + stride]);
    }
    __syncthreads();
  }

  if (tid == 0) dst[blockIdx.x] = vec[0];
}

constexpr int PACKING_SIZE = 32;
__global__ void packing(int* data, size_t chunk_size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= chunk_size) return;
#pragma unroll
  for (int i = 1; i < PACKING_SIZE; ++i) {
    data[idx] <<= 1;
    data[idx] |= data[i * chunk_size + idx] & 0x01;
  }
}

__global__ void unpacking(float* dst, const int* src, size_t chunk_size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= chunk_size) return;
  unsigned int mask = 1;
#pragma unroll
  for (int i = PACKING_SIZE - 1; i >= 0; --i) {
    int sign_bit = (src[idx] & mask) >> (PACKING_SIZE - i - 1);
    int sign = -((sign_bit << 1) - 1);
    dst[i * chunk_size + idx] = sign;
    mask <<= 1;
  }
}

namespace byteps {
namespace common {
constexpr int BLOCK_PER_GRID = 1024;
// int CpuReducer::sum(void* dev_dst, const void* dev_src, size_t len, int
// dtype,
//                     float alpha) {
//   int thread_per_block = ((len/4) + BLOCK_PER_GRID - 1) / BLOCK_PER_GRID;
//   sum_kernel<<<BLOCK_PER_GRID, thread_per_block>>>(
//       reinterpret_cast<float*>(dev_dst),
//       reinterpret_cast<const float*>(const_cast<void*>(dev_src)), len / 4,
//       alpha);
//   return 0;
// }

int CpuReducer::sum(void* dev_dst, const void* dev_src1, const void* dev_src2,
                    size_t len, int dtype, float alpha) {
  int thread_per_block = ((len / 4) + BLOCK_PER_GRID - 1) / BLOCK_PER_GRID;
  sum_kernel<<<BLOCK_PER_GRID, thread_per_block>>>(
      reinterpret_cast<float*>(dev_dst),
      reinterpret_cast<const float*>(const_cast<void*>(dev_src1)),
      reinterpret_cast<const float*>(const_cast<void*>(dev_src2)), len / 4,
      alpha);
  return 0;
}

int CpuReducer::sign(void* dev_dst, const void* dev_src, size_t len,
                     int dtype) {
  int thread_per_block = ((len / 4) + BLOCK_PER_GRID - 1) / BLOCK_PER_GRID;
  sign_kernel<<<BLOCK_PER_GRID, thread_per_block>>>(
      reinterpret_cast<int*>(dev_dst),
      reinterpret_cast<const float*>(const_cast<void*>(dev_src)), len / 4);
  return len / 4;
}

float CpuReducer::norm1(const void* dev_src, void* dev_dst, void* dst,
                        size_t len, int dtype) {
  int x = ((len / 4) + BLOCK_PER_GRID - 1) / BLOCK_PER_GRID;
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  ++x;

  norm1_kernel<<<BLOCK_PER_GRID, x>>>(
      reinterpret_cast<const float*>(const_cast<void*>(dev_src)),
      reinterpret_cast<float*>(dev_dst), len / 4);

  hipMemcpy(dst, dev_dst, BLOCK_PER_GRID * 4, hipMemcpyDeviceToHost);

  float ret = 0;
  auto p_dst = reinterpret_cast<float*>(dst);
  for (int i = 0; i < BLOCK_PER_GRID; ++i) {
    ret += p_dst[i];
  }

  return ret;
}

namespace compressor {

size_t OnebitCompressor::PackingCuda(void* data, size_t len, int dtype) {
  size_t padding_len = (PACKING_SIZE - (len % PACKING_SIZE)) % PACKING_SIZE;
  size_t chunk_size = (len + padding_len) / PACKING_SIZE;

  int thread_per_block = (chunk_size + BLOCK_PER_GRID - 1) / BLOCK_PER_GRID;
  packing<<<BLOCK_PER_GRID, thread_per_block>>>(reinterpret_cast<int*>(data),
                                                chunk_size);
  return chunk_size * 4;
}

size_t OnebitCompressor::UnpackingCuda(void* dst, const void* src, size_t len,
                                       int dtype) {
  auto chunk_size = (len - sizeof(float)) / 4;
  int thread_per_block = (chunk_size + BLOCK_PER_GRID - 1) / BLOCK_PER_GRID;
  unpacking<<<BLOCK_PER_GRID, thread_per_block>>>(
      reinterpret_cast<float*>(dst),
      reinterpret_cast<const int*>(const_cast<void*>(src)), chunk_size);
  return chunk_size;
}
}  // namespace compressor
}  // namespace common
}  // namespace byteps
